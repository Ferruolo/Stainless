#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <iostream>
#include "library.cuh"

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define BLOCKSIZE 32
//Helpers and Kernels


//Return -> size
//Modify -> newShapeArray
//Expect -> num_dims, oldShapeArray
int copyShape(const int &num_dim, int *oldShape, int * new_shape) {

    int size = 1;
    for (int _i = 0; _i < num_dim; ++_i){
        new_shape[_i] = oldShape[_i];
        size *= new_shape[_i];
    }
    return size;
}

void hello() {
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(NULL);
    cudaHello<<<1, 1>>>();
}


struct Matrix * MatrixFactory(const int *shape, int num_dum, location loc){
    if (loc != GPU){
        std::cerr << "Only supports GPU atm" << std::endl;
        exit(1);
    }
    if (!shape || num_dum > 2) {
        std::cerr << "Shape incorrectly specified" << std::endl;
    }
    Matrix * m = (Matrix *) malloc(sizeof(Matrix));

    int *new_shape = (int *) malloc(num_dum * sizeof(int));
    for (int _i = 0; _i < num_dum; ++_i){
        new_shape[_i] = shape[_i];
    }

    //Only supports 2-dimensional at the moment
    int size = new_shape[0] * new_shape[1];

    float *elements;

    hipMalloc(&elements, sizeof(float) * size);


    m->elements = elements;
    m->size = size;
    m->shape = new_shape;
    m->num_dim = num_dum;
    m->loc = loc;
    return m;
}


Matrix * CreateUniformRandomMatrix(const int * shape, int num_dim, location loc, int min_val, int max_val) {
    Matrix * m = MatrixFactory(shape, num_dim, loc);
    dim3 gridDim(CEIL_DIV(m->size, BLOCKSIZE * BLOCKSIZE));
    int blockThreads = min(BLOCKSIZE*BLOCKSIZE, m->size);
    dim3 blockDim(blockThreads);

    cuRandArrInit<<<gridDim, blockDim>>>(m->elements, min_val, max_val, m->size);
    return m;
}

struct Matrix *CreateConstMatrix(int num_dim, const int *shape, int c, location loc) {
    Matrix * m = MatrixFactory(shape, num_dim, loc);
    dim3 gridDim(CEIL_DIV(m->size, BLOCKSIZE * BLOCKSIZE));
    dim3 blockDim(BLOCKSIZE * BLOCKSIZE);
    cuConstArrInit<<<gridDim, blockDim>>>(m->elements, m->size, c);
    return m;
}

struct Matrix *CreateZeroMatrix(int num_dim, const int *shape, location loc) {
    return CreateConstMatrix(num_dim, shape, 0, loc);
}

void printMatrix(const struct Matrix *m) {
    float *temp = (float *) malloc(m->size * sizeof(float));
    hipMemcpy(temp, m->elements, m->size * sizeof(float), hipMemcpyDeviceToHost);
    for (int _i = 0; _i < m->shape[0]; ++_i){
        for (int j = 0; j < m->shape[1]; ++j) {
            printf("%.2f ", temp[_i * m->shape[1] + j]);
        }
        printf("\n");
    }
    printf("\n");
    free(temp);
}

int checkMatrixEquality(const struct Matrix *m1, const struct Matrix *m2) {
    if (m1->num_dim != m2->num_dim) {
        return false;
    }
    for (int _i = 0; _i < m1->num_dim; ++_i) {
        if (m1->shape[_i] != m2->shape[_i]) return false;
    }
    bool * equalityChecker;
    hipMallocManaged(&equalityChecker, sizeof(bool) * m1->size);
    dim3 gridDim(CEIL_DIV(m1->size, BLOCKSIZE * BLOCKSIZE));
    int blockThreads = min(BLOCKSIZE*BLOCKSIZE, m1->size);
    dim3 blockDim(blockThreads);
    checkEqualityKernel<<<gridDim, blockDim>>>(m1->elements,
                                               m2->elements,
                                               equalityChecker,
                                               m1->size);

    hipDeviceSynchronize();
    for (int _i = 0; _i < m1->size; ++_i) {
        if (!equalityChecker[_i]) {
            printf("issue at %d\n", _i);
            hipFree(equalityChecker);
            return false;
        }
    }

    hipFree(equalityChecker);
    return true;
}

struct Matrix * MatMul(const struct Matrix *a, const struct Matrix *b) {
    if (a->shape[1] != b->shape[0]) {
        printf("Matrix size mismatched");
        exit(1);
    }
    int new_shape[2] = {a->shape[0], b->shape[1]};

    Matrix *C = CreateZeroMatrix(b->num_dim, new_shape, GPU);

    dim3 gridDim(CEIL_DIV(new_shape[0], BLOCKSIZE), CEIL_DIV(new_shape[1], BLOCKSIZE), 1);

    dim3 blockDim(BLOCKSIZE, BLOCKSIZE);

    sgemm_kernel<<<gridDim, blockDim>>>(a->shape[0],
            b->shape[1],
            b->shape[0],
            1.0, 0.0, a->elements, b->elements, C->elements);
    hipDeviceSynchronize();
    return C;
}