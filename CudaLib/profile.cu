#include "library.cuh"
#include "kernels.cuh"
#include "hipblas.h"

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K, &alpha,
                reinterpret_cast<const float*>(A), M,
                reinterpret_cast<const float*>(B), K, &beta,
                reinterpret_cast<float*>(C), M);

    hipblasDestroy(handle);
    return 0;
}



